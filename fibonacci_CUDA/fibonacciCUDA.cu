#include "stdio.h" 
#include "stdlib.h" 
#include "time.h"  
#include "math.h"
#include "hip/hip_runtime.h"


#include <memory> 
__device__ void fibonacci(int number, int* res);
__global__ void fibonacciParent(int* number, int* res);

int initial_fibonacci_run(int fib, int currentDepth, int targetDepth);

int getSPcores(hipDeviceProp_t devProp)
{
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    printf("%d   %d   %d   %s\n", devProp.major, devProp.minor, mp, devProp.name);
    switch (devProp.major){
    case 2: // Fermi
        if (devProp.minor == 1) cores = mp * 48;
        else cores = mp * 32;
        break;
    case 3: // Kepler
        cores = mp * 192;
        break;
    case 5: // Maxwell
        cores = mp * 128;
        break;
    case 6: // Pascal
        if (devProp.minor == 1) cores = mp * 128;
        else if (devProp.minor == 0) cores = mp * 64;
        else printf("Unknown device type\n");
        break;
    case 7: //GeForce GTX 1650
        cores = mp * 64;
        break;
    default:
        printf("Unknown device type\n");
        break;
    }
    return cores;
}

int calc_CUDA_Fibonacci(int number){
    //get number of cores

    // The number of Graphics cards in this computer
    int deviceCount = 0;
    // The Index Device we're going to use (Default 0)
    int currentDevice = 0;
    // The number of CUDA cores on the device being used
    int CUDACoreCount = 0;
    // How deep we need to go into recursion to get the optimal number of threads.
    int depth = 0;
    // Error logging
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    if (deviceCount == 0){
        printf("Cannot find any CUDA device.");
        exit(EXIT_FAILURE);
    }

    // Get the information on the current device
    hipSetDevice(currentDevice);
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, currentDevice);

    // Get the number of CUDA Cores on the current device
    CUDACoreCount = getSPcores(deviceProperties);

    // How deep we need to go into the recursion before we spawn threads
    depth = floor(log2((double)CUDACoreCount));
    // printf("   \n%d\n   ",depth);
    int x = initial_fibonacci_run(number, 0, depth);   
    return x;


}


int main(int argc, char* argv[]){
    if(argc != 2){
        printf("Error: You have to insert the input number for fibonacci!\n");
        return 0;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("The input number for fibonacci is: %s\n", argv[1]);

    int number = atoi(argv[1]);
    int res;
    float elapsedTime;
    
    hipEventRecord(start, 0);

    printf("The fibonacci of %s is %d\n", argv[1], calc_CUDA_Fibonacci(number));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);    
    printf("Fibonacci computation in %f seconds\n",elapsedTime/1000);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int initial_fibonacci_run(int fib, int currentDepth, int targetDepth){
    if (fib <= 1)
        return fib;

    if (currentDepth < targetDepth)
        return initial_fibonacci_run(fib - 1, currentDepth++, targetDepth) + initial_fibonacci_run(fib - 2, currentDepth++, targetDepth);

    int *d_fib, *d_result;

    int size = sizeof(int);
    hipMalloc((void**)&d_fib, size);
    hipMalloc((void**)&d_result, size);

    hipMemcpy(d_fib, &fib, size, hipMemcpyHostToDevice);
    hipMemcpy(d_result, 0, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 64;
    int blocksPerGrid = (fib + threadsPerBlock - 1) / threadsPerBlock;
    fibonacciParent<<<blocksPerGrid, threadsPerBlock>>>(d_fib, d_result);
    int result = 0;
    hipMemcpy(&result, d_result, size, hipMemcpyDeviceToHost);    

    hipFree(d_fib);
    hipFree(d_result);
    return result;
}

__global__ void fibonacciParent(int* number, int* res){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < *number){
        fibonacci(*number, res);
    }
   
    return;

}

__device__ void fibonacci(int number, int* res){

    if (number <= 1){
        *res += number;
        return;
    }

    int i =  blockIdx.x * blockDim.x + threadIdx.x;

        fibonacci(number - 1, res);
        fibonacci(number - 2, res);
}
